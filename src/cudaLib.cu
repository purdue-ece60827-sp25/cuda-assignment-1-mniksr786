#include "hip/hip_runtime.h"

#include "cudaLib.cuh"
#include "../src/cpuLib.cpp"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size) {
		y[i] = scale * x[i] + y[i];
		}	
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n\n";
	std::cout << "My First GPU Kernel!\n";
	std::cout << "Many more to come!\n";
	
	auto tStart = std::chrono::high_resolution_clock::now();
	//	Insert code here
	
	float *x, *y, *z;
	float *x_d, *y_d, *z_d;
	float a = 2.0f;
	int vector_mem = vectorSize * sizeof(float);
	
	// Space allocation for the float input vectors
	x = (float *) malloc(vector_mem);
	y = (float *) malloc(vector_mem);
	z = (float *) malloc(vector_mem);
	
	// Creating the vectors using the CPU functions only
	vectorInit(x, vectorSize);
	vectorInit(y, vectorSize);
	
	
	// Allocating space and copying data to the GPU
	hipMalloc((void **) &x_d, vector_mem);
	hipMalloc((void **) &y_d, vector_mem);
	hipMalloc((void **) &z_d, vector_mem);
	
	hipMemcpy(x_d, x, vector_mem, hipMemcpyHostToDevice);
	hipMemcpy(y_d, y, vector_mem, hipMemcpyHostToDevice);
	hipMemcpy(z_d, y, vector_mem, hipMemcpyHostToDevice);
	
	
	// Calling the SAXPY kernel
	saxpy_gpu<<< (vectorSize/256), 256 >>>(x_d, z_d, a, vectorSize);
	
	// Transfering data back from GPU to CPU
	hipMemcpy(z, z_d, vector_mem, hipMemcpyDeviceToHost);
	hipMemcpy(y, y_d, vector_mem, hipMemcpyDeviceToHost);
	hipMemcpy(x, x_d, vector_mem, hipMemcpyDeviceToHost);

	// Free GPU memory
	hipFree(x_d); hipFree(y_d); hipFree(z_d);
		
	// Verifying the error count
	int errorCount = verifyVector(x, y, z, a, vectorSize);
	std::cout << "Found " << errorCount << " / " << vectorSize << " errors \n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

/* 
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is responsible for sampleSize points. 
 *pSums is a pointer to an array that holds the number of 'hit' points for each thread. The length of this array is pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

__global__
void generatePoints (uint64_t *pSums, uint64_t *pSumSize, uint64_t sampleSize) {
	//	Insert code here
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	float x, y;
	uint64_t hits =0;
	
	hiprandState_t rng;
	hiprand_init(clock64(), i, 0, &rng);    
    
	for (uint64_t idx = 0; idx < sampleSize; ++idx) {
		x = hiprand_uniform(&rng);
		y = hiprand_uniform(&rng);

		if ((x * x + y * y) <= 1.0f ) {
			hits++;
		}
	}
	atomicAdd(reinterpret_cast<unsigned long long*>(pSums), hits);
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	std::string str;
	uint64_t hitCount = 0;
	uint64_t *hitCount_d, *totalHitCount_d;
	
	auto tStart = std::chrono::high_resolution_clock::now();
	
	hipMalloc((void **) &hitCount_d, sizeof(uint64_t));
	//hipMalloc((void **) &totalHitCount_d, sizeof(uint64_t));
	
	hipMemcpy(hitCount_d, &hitCount, sizeof(uint64_t), hipMemcpyHostToDevice);
	//hipMemcpy(totalHitCount_d, &totalHitCount, sizeof(uint64_t), hipMemcpyHostToDevice);

	//	Main GPU Monte-Carlo Code
	generatePoints<<<(generateThreadCount/128), 128>>>(hitCount_d, totalHitCount_d, sampleSize);
	
	hipMemcpy(&hitCount, hitCount_d, sizeof(uint64_t), hipMemcpyDeviceToHost);
	//hipMemcpy(&totalHitCount, totalHitCount_d, sizeof(uint64_t), hipMemcpyDeviceToHost);

	hipFree(hitCount_d); 
	//hipFree(totalHitCount_d);
	
	
	float approxPi = (((double)hitCount) / (sampleSize * generateThreadCount));
	approxPi = approxPi * 4.0f;
		
	std::cout << std::setprecision(10);
	std::cout << "Estimated Pi = " << approxPi << "\n";
		
	/* float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";*/

	auto tEnd= std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 0;

	//      Insert code here
	// NOT USING THIS FUNCTION
	std::cout << "Sneaky, you are ...\n";
	std::cout << "Compute pi, you must!\n";
	return approxPi;
}
